#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "BasicGPU.cuh"

__global__ void gameOfLifeKernel(const unsigned char* inputGrid, unsigned char* outputGrid, int width, int height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        size_t size = width * height;
        unsigned int index = x + y * width;
        unsigned int x1 = index % width;
        unsigned int y1 = index - x1;
        unsigned int y0 = (y1 + size - width) % size;
        unsigned int y2 = (y1 + width) % size;
        unsigned int x0 = (x1 + width - 1) % width;
        unsigned int x2 = (x1 + 1) % width;
        unsigned char aliveCells = inputGrid[x0 + y0] + inputGrid[x1 + y0] + inputGrid[x2 + y0] + inputGrid[x0 + y1] + inputGrid[x2 + y1] + inputGrid[x0 + y2] + inputGrid[x1 + y2] + inputGrid[x2 + y2];
        outputGrid[index] = aliveCells == 3 || (aliveCells == 2 && inputGrid[index]) ? 1 : 0;
    }
}

void runCudaSimulation(unsigned int width, unsigned int height, unsigned char* m_data, unsigned char* d_data, unsigned char* d_resultData, size_t dataBytes) {
    dim3 nThreads(32, 32);
    dim3 nBlocks((width + nThreads.x - 1) / nThreads.x, (height + nThreads.y - 1) / nThreads.y);
    gameOfLifeKernel <<<nBlocks, nThreads>>> (d_data, d_resultData, width, height);
    hipDeviceSynchronize();
    if (hipMemcpy(m_data, d_resultData, dataBytes, hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "Error copying data from device to host" << std::endl;
	}
}